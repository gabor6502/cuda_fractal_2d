#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
//#include "cuda_gl_interop.h"
#include "kernel.h"

// warnings for floats being used in cuda/std/complex, just redefine infinity
#define INFINITY std::numeric_limits<double>::infinity()
#include <cuda/std/complex>

#include <stdlib.h>
#include <stdio.h>

// the standard cuda return checking macro
#define CUDA_CHECK_RETURN(value)                                     \
{                                                                    \
    hipError_t _m_cudaStat = value;                                 \
    if ( _m_cudaStat != hipSuccess)                                 \
    {                                                                \
        fprintf(stderr, "Error %s at line %d in file %s\n",          \
            hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);    \
        exit(EXIT_FAILURE);                                          \
    }                                                                \
} 

typedef cuda::std::complex<double> complex;

struct DeviceParams
{
    int max_block_size_x;
    int max_block_size_y;

    int max_grid_size_x;
    int max_grid_size_y;
};
DeviceParams deviceParams;

struct MandelbrotImageParams
{
    int width;
    int height;

    int iterations;
};
MandelbrotImageParams imageParams;

struct HostBuffers
{
    unsigned int * h_dwell_map;
    float * h_image_colours;
};
HostBuffers hostBuffers;

struct DeviceBuffers
{
    unsigned int * d_dwell_map;
    float * d_image_colours;

    //unsigned int pbo; // hanlde to the pbo GLuint
    //hipGraphicsResource** image_resource; // links PBO and device buffer d_image_colours
};
DeviceBuffers deviceBuffers;


// starting off with basic alg, no DP/mariani-silver yet

__device__ int mandelbrot_pixel(int max_iterations,
                                int width_dwell_map, int height_dwell_map,
                                complex bottom_left, complex top_right,
                                int pixel_x, int pixel_y)
{
    // convert from image-segment space to mandelbrot space
    complex dist_max_min = top_right - bottom_left;
    complex c = bottom_left + complex((float)pixel_x / (float)width_dwell_map * dist_max_min.real(),
        (float)pixel_y / (float)height_dwell_map * dist_max_min.imag());
    complex z = c; // iterate starting at c

    int dwell = 0;
    while (dwell++ < max_iterations && z.real() * z.real() + z.imag() * z.imag() < MAX_MAG_SQ)
    {
        z = z * z + c;
    }

    return dwell;
}

__global__ void k_mandelbrot(unsigned int * d_dwell_map, int max_iterations,
                                   int width_dwell_map, int height_dwell_map, 
                                   complex bottom_left, complex top_right)
{
    int img_x = threadIdx.x + blockDim.x * blockIdx.x;
    int img_y = threadIdx.y + blockDim.y * blockIdx.y;

    if (img_x < width_dwell_map && img_y < height_dwell_map)
    {
        d_dwell_map[img_y * height_dwell_map + img_x] 
            = mandelbrot_pixel(max_iterations, width_dwell_map, height_dwell_map, 
                               bottom_left, top_right, 
                                img_x, img_y);
    }

}

// assign colours of the screen from dwell-map

__global__ void k_colour_dwell_map(float * d_pixels, unsigned int * d_dwell_map, int max_iterations, int width_dwell_map, int height_dwell_map)
{
    int img_x = threadIdx.x + blockDim.x * blockIdx.x;
    int img_y = threadIdx.y + blockDim.y * blockIdx.y;

    int dwell;

    if (img_x < width_dwell_map && img_y < height_dwell_map)
    {
        dwell = d_dwell_map[img_y * width_dwell_map + img_x];

        if (dwell < max_iterations)
        {
            // temp, just set white tone for now
            d_pixels[img_y * width_dwell_map + img_x] = (float)dwell / (float)max_iterations;;
        }
        else
        {
            d_pixels[img_y * width_dwell_map + img_x] = 0.0f;
        }
    }
}

// -- functions to allow access to CUDA from app --

void setImageSize(int w, int h)
{
    imageParams.width = w;
    imageParams.height = h;
}

void setIterations(int iter)
{
    imageParams.iterations = iter;
}

void initCUDA()
{

    int deviceCount = 0;
    hipDeviceProp_t deviceProp;

    printf("Initializing CUDA ... ");

    // make sure the hardware is CUDA compatible
    CUDA_CHECK_RETURN(hipGetDeviceCount(&deviceCount));

    if (deviceCount == 0)
    {
        printf("\nCUDA is not supported on this machine!\n");
        exit(EXIT_FAILURE);
    }

    CUDA_CHECK_RETURN(hipSetDevice(DEVICE));

    CUDA_CHECK_RETURN(hipGetDeviceProperties(&deviceProp, DEVICE));

    deviceParams.max_block_size_x = deviceProp.maxThreadsDim[0];
    deviceParams.max_block_size_y = deviceProp.maxThreadsDim[1];

    deviceParams.max_grid_size_x = deviceProp.maxGridSize[0];
    deviceParams.max_grid_size_y = deviceProp.maxGridSize[1];

    printf("done.\n");
}
/*
void initCudaOpenGLInterop(unsigned int pbo)
{
    size_t device_buffer_size = sizeof(unsigned int) * imageParams.width * imageParams.height;

    deviceBuffers.pbo = pbo;

    CUDA_CHECK_RETURN(cudaGLSetGLDevice(DEVICE));

    // register the pbo under the resource given (r/w flag assumed)
    CUDA_CHECK_RETURN(
        hipGraphicsGLRegisterBuffer(deviceBuffers.image_resource, pbo, hipGraphicsRegisterFlagsNone));

    // map the pbo resource pointer to the device buffer supplied
    CUDA_CHECK_RETURN(
        hipGraphicsResourceGetMappedPointer((void **)deviceBuffers.d_image_colours, 
                                              &device_buffer_size,
                                              *deviceBuffers.image_resource));
}*/

void allocCUDA()
{
    printf("Allocating host memory ... ");
    hostBuffers.h_dwell_map = new unsigned int[imageParams.width * imageParams.height];
    hostBuffers.h_image_colours = new float[imageParams.width * imageParams.height]; // black and white, for now
    printf("done.\n");

    printf("Allocating device memory ... ");
    CUDA_CHECK_RETURN(
        hipMalloc((void**) &deviceBuffers.d_dwell_map, sizeof(unsigned int) * imageParams.width * imageParams.height)
    );
    CUDA_CHECK_RETURN(
        hipMalloc((void**) &deviceBuffers.d_image_colours, sizeof(unsigned int) * imageParams.width * imageParams.height) // black and white, for now
    );
    printf("done.\n");
}

void runCUDA()
{
    const unsigned int nThrdX = ceil((float) imageParams.width / (float) deviceParams.max_block_size_x);
    const unsigned int nThrdY = ceil((float) imageParams.height / (float) deviceParams.max_block_size_y);
    dim3 blockDimensions = dim3(nThrdX, nThrdY);

    const unsigned int nBlkX = ceil((float) nThrdX / (float)deviceParams.max_grid_size_x);
    const unsigned int nBlkY = ceil((float) nThrdY / (float)deviceParams.max_grid_size_y);
    dim3 gridDimensions = dim3(nBlkX, nBlkY);

    complex bottom_left(-1.5, 1.0);
    complex top_right(0.5, 1.0);

    printf("Executing kernels ... ");

    k_mandelbrot<<<blockDimensions, gridDimensions>>>(deviceBuffers.d_dwell_map, imageParams.iterations, imageParams.width, imageParams.height, bottom_left, top_right);

    k_colour_dwell_map<<<blockDimensions, gridDimensions>>>(deviceBuffers.d_image_colours, deviceBuffers.d_dwell_map, imageParams.iterations, imageParams.width, imageParams.height);

    printf("done!\n");
}

void deallocCUDA()
{
    printf("Deallocating device memory ... ");
    CUDA_CHECK_RETURN(hipFree((void*) deviceBuffers.d_dwell_map));
    CUDA_CHECK_RETURN(hipFree((void*) deviceBuffers.d_image_colours));
    CUDA_CHECK_RETURN(hipDeviceReset());
    printf("done.\n");

    printf("Deallocating host memory ... ");
    delete[] hostBuffers.h_dwell_map;
    delete[] hostBuffers.h_image_colours;
    printf("done.\n");
}
