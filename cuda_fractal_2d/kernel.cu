﻿#include "kernel.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdlib.h>
#include <stdio.h>

// the standard cuda return checking macro
#define CUDA_CHECK_RETURN(value)                                     \
{                                                                    \
    hipError_t _m_cudaStat = value;                                 \
    if ( _m_cudaStat != hipSuccess)                                 \
    {                                                                \
        fprintf(stderr, "Error %s at line %d in file %s\n",          \
            hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);    \
        exit(EXIT_FAILURE);                                          \
    }                                                                \
} 

void initCUDA()
{
    printf("Initializing CUDA ... ");

    // make sure the hardware is CUDA compatible
    int deviceCount = 0;
    CUDA_CHECK_RETURN(hipGetDeviceCount(&deviceCount));

    if (deviceCount == 0)
    {
        printf("\nCUDA is not supported on this machine!\n");
        exit(EXIT_FAILURE);
    }

    CUDA_CHECK_RETURN(hipSetDevice(0)); // use the first device available

    printf("done.\n");
}


void runCUDA()
{
    
}

void deallocCUDA()
{
    CUDA_CHECK_RETURN(hipDeviceReset());
}
